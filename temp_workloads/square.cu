#include <hip/hip_runtime.h>
#include <iostream>

__global__ void square_array(int *a, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        a[idx] = a[idx] * a[idx];
    }
}

int main() {
    int n = 100;
    int *a, *d_a;
    hipMalloc(&d_a, n * sizeof(int));
    a = new int[n];

    // Initialize the input array with random values
    for (int i = 0; i < n; i++) {
        a[i] = rand() % 100;
    }

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel on the GPU with high occupancy
    for(int i=0;i<=5000000;i++)
    {
        square_array<<<(n + 255) / 256, 256>>>(d_a, n);
    }
    
    hipMemcpy(a, d_a, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print the results
    // for (int i = 0; i < n; i++) {
    //     std::cout << a[i] << " ";
    // }

    hipFree(d_a);
    delete[] a;
    return 0;
}
