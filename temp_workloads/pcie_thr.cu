#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 512

__global__ void kernel(float *d_out, float *d_in) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = d_in[i];
}

int main() {
    float *h_in, *h_out;
    float *d_in, *d_out;
    int size = 1 << 15; // 1MB
     printf("%d\n",size);

    hipHostMalloc(&h_in, size, hipHostMallocDefault);
    hipHostMalloc(&h_out, size, hipHostMallocDefault);
    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);

    // Initialize host memory
    for (int i = 0; i < size; i++) {
        h_in[i] = (float)i;
        // printf("%d\n",i);
        
    }
    // printf("hello\n");
    // Copy host memory to device memory
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    // Launch kernel
    kernel<<<size/BLOCK_SIZE, BLOCK_SIZE>>>(d_out, d_in);

    // Copy device memory to host memory
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    // cudaFreeHost(h_in);
    // cudaFreeHost(h_out);

    return 0;
}
